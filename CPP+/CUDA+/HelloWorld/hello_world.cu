#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to print "Hello, World!" from each thread
__global__ void helloWorldKernel() {
    printf("Hello, World! from thread [%d, %d]\n", blockIdx.x, threadIdx.x);
}

int main() {
    // Launch the kernel with 1 block and 5 threads
    helloWorldKernel<<<1, 5>>>();

    // Wait for the GPU to finish before accessing the results
    hipDeviceSynchronize();

    return 0;
}
